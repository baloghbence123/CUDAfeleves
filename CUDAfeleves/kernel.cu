#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <malloc.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <stdio.h>
using namespace std;

#define IMG_INPUT "C:\\Users\\Bence\\source\\repos\\CUDAfeleves\\Kepek\\input1.bmp"

#define IMG_OUTPUT "C:\\Users\\Bence\\source\\repos\\CUDAfeleves\\Kepek\\output.bmp"
#define IMG_OUTPUT2 "C:\\Users\\Bence\\source\\repos\\CUDAfeleves\\Kepek\\output2.bmp"

//4000*4000
//3613*5420
#define IMG_HEADER 1078
#define IMG_WIDTH 4000
#define IMG_HEIGHT 4000
#define Channel 3
	
#define ImgSize (IMG_HEADER+IMG_WIDTH*IMG_HEIGHT*Channel)
//a csík issue a dimSize-al lesz kapcsolatban
#define dimSize  32
#define FilterSize 3
#define tileSize (dimSize - ((FilterSize/2)*2))

__device__ unsigned char dev_imgin[IMG_HEADER + IMG_WIDTH * IMG_HEIGHT * Channel];
__device__ unsigned char dev_img_result[IMG_HEADER + IMG_WIDTH * IMG_HEIGHT * Channel];

__constant__ int dev_filter[FilterSize*FilterSize];

__global__ void GrayPicMultipleBlocks() {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int actual = IMG_WIDTH * Channel * y + Channel * x;


	if (IMG_HEADER + actual+Channel-1<ImgSize)
	{
		int tmp = 0;
		for (int i = 0; i < Channel; i++)
		{
			tmp += (int)dev_imgin[IMG_HEADER + actual + i];
		}
		tmp /= Channel;
		for (int i = 0; i < Channel; i++)
		{
			dev_img_result[IMG_HEADER + actual + i]=(unsigned char)tmp;
		}

		

	}
	//if (true)
	//{
	//	int tmp = 0;
	//	tmp += (int)dev_img[IMG_HEADER + actualr];
	//	tmp += (int)dev_img[IMG_HEADER + actualg];
	//	tmp += (int)dev_img[IMG_HEADER + actualb];
	//	tmp /= 3;
	//	dev_img[IMG_HEADER + actualr]= (unsigned char)tmp;
	//	dev_img[IMG_HEADER + actualg]= (unsigned char)tmp;
	//	dev_img[IMG_HEADER + actualb]= (unsigned char)tmp;
	//}


}



__global__ void AnyFilter() {


	__shared__ int shr_Filter[FilterSize][FilterSize];
	if ((threadIdx.x<FilterSize) && (threadIdx.y<FilterSize))
	{
		shr_Filter[threadIdx.y][threadIdx.x] = dev_filter[threadIdx.y*FilterSize+threadIdx.x];
	}
	__syncthreads();

	__shared__ unsigned char shr_CMatrix[dimSize][dimSize * Channel];
	


	

		int xout = (tileSize * blockIdx.x) + threadIdx.x;
		int yout = (tileSize * blockIdx.y) + threadIdx.y;

		if (yout < IMG_HEIGHT && xout < IMG_WIDTH)
		{

			int ymatrix = yout - (FilterSize / 2);
			int xmatrix = xout - (FilterSize / 2);

			//int actual = IMG_WIDTH * Channel * y_i + Channel * x_i;


			if ((0 <= ymatrix) && (ymatrix < IMG_HEIGHT) && (0 <= xmatrix) && (xmatrix < IMG_WIDTH))
			{
				for (int i = 0; i < Channel; i++)
				{


					shr_CMatrix[threadIdx.y][(threadIdx.x * Channel) + i] = dev_imgin[IMG_HEADER + (IMG_WIDTH * Channel * ymatrix) + (Channel * xmatrix) + i];
				}
			}
			else
			{
				//Feltoltes az utolso ertekkel
				if (0>ymatrix)
				{
					for (int i = 0; i < Channel; i++)
					{
						int actual = IMG_HEADER + (IMG_WIDTH * Channel * 0) + (Channel * xmatrix) + i;
						shr_CMatrix[threadIdx.y][(threadIdx.x * Channel) + i] = dev_imgin[actual];

					}
				}
				else if (IMG_HEIGHT<=ymatrix)
				{
					for (int i = 0; i < Channel; i++)
					{
						int actual = IMG_HEADER + (IMG_WIDTH * Channel * (IMG_HEIGHT-1)) + (Channel * xmatrix) + i;
						shr_CMatrix[threadIdx.y][(threadIdx.x * Channel) + i] = dev_imgin[actual];

					}
				}
				else if (0>xmatrix)
				{
					for (int i = 0; i < Channel; i++)
					{
						int actual = IMG_HEADER + (IMG_WIDTH * Channel * ymatrix) + (Channel * 0) + i;
						shr_CMatrix[threadIdx.y][(threadIdx.x * Channel) + i] = dev_imgin[actual];

					}
				}
				else if (IMG_WIDTH<=xmatrix)
				{
					for (int i = 0; i < Channel; i++)
					{
						int actual = IMG_HEADER + (IMG_WIDTH * Channel * ymatrix) + (Channel * IMG_WIDTH-1) + i;
						shr_CMatrix[threadIdx.y][(threadIdx.x * Channel) + i] = dev_imgin[actual];

					}
				}

				//feltoltes c szammal
				//for (int i = 0; i < Channel; i++)
				//{
				//	shr_CMatrix[threadIdx.y][(threadIdx.x * Channel) + i] = 0;
				//	//shr_CMatrix[threadIdx.y][(threadIdx.x * Channel) + i] = 1;
				//}
			}


			__syncthreads();

			int sum[3] = { 0,0,0 };

			if (threadIdx.x < tileSize && threadIdx.y < tileSize)
			{
				for (int i = 0; i < FilterSize; i++)
				{
					for (int j = 0; j < FilterSize; j++)
					{
						for (int c = 0; c < Channel; c++)
						{
							int tmp = ((int)shr_CMatrix[threadIdx.y + i][((threadIdx.x + j) * Channel) + c] * shr_Filter[i][j]);
							sum[c] += tmp;


							//sum[c] += (int)shr_CMatrix[threadIdx.y + i][((threadIdx.x + j) * Channel) + c];


						}

					}
				}
			}

			//Blur effect-hez
			//for (size_t i = 0; i < Channel; i++)
			//{
			//	sum[i] /= 256;
			//}
			
			

			if (threadIdx.x < tileSize && threadIdx.y < tileSize)
			{

				for (int i = 0; i < Channel; i++)
				{
					int actual = IMG_HEADER + (IMG_WIDTH * Channel * yout) + (Channel * xout) + i;

					dev_img_result[actual] = (unsigned char)sum[i];
				}
			}
		}

	




}




int main()
{

	unsigned char* img;
	unsigned char* host_image;
	FILE* f_input_img, * f_output_img;
	//int host_filter[FilterSize * FilterSize] = {1,4,6,4,1,4,16,24,16,4,6,24,36,24,6,4,16,24,16,4,1,4,6,4,1};
	
	//int host_filter[FilterSize * FilterSize] = { 1, 0, 1, 0,5,0 , 1,0,1 };
	//int host_filter[FilterSize * FilterSize] = { 0, -1, 0, -1,4,-1 ,0,-1,0 };
	int host_filter[FilterSize * FilterSize] = { -1, -1, -1, -1,8,-1 , -1,-1,-1 };
	//int host_filter[FilterSize * FilterSize] = { 0, 0, 0, 1, 0, 0, 0,0,0};
	// 
	// 
	// 
	// Load image

	img = (unsigned char*)malloc(IMG_HEADER + sizeof(unsigned char) * IMG_WIDTH * IMG_HEIGHT * Channel);
	host_image = (unsigned char*)malloc(IMG_HEADER + sizeof(unsigned char) * IMG_WIDTH * IMG_HEIGHT * Channel);

	fopen_s(&f_input_img, IMG_INPUT, "rb");
	fread(img, 1, IMG_HEADER + IMG_WIDTH * IMG_HEIGHT * Channel, f_input_img);
	fclose(f_input_img);
	unsigned int tmpInt[IMG_HEADER];
	unsigned char tmpSet[IMG_HEADER];


	hipMemcpyToSymbol(HIP_SYMBOL(dev_filter), host_filter, FilterSize * FilterSize * sizeof(int));

	//ToDo: Memory copy H->D
	hipMemcpyToSymbol(HIP_SYMBOL(dev_imgin), img, IMG_HEADER + IMG_WIDTH * IMG_HEIGHT * Channel * sizeof(unsigned char));

	hipMemcpyToSymbol(HIP_SYMBOL(dev_img_result), img, IMG_HEADER * sizeof(unsigned char));
	//hipMemcpyToSymbol(HIP_SYMBOL(dev_img_result), img, IMG_HEADER + IMG_WIDTH * IMG_HEIGHT * Channel * sizeof(unsigned char));

	//hipMemcpyToSymbol(HIP_SYMBOL(dev_filter), host_filter, FilterSize * FilterSize * sizeof(int));



	dim3 grid_size((IMG_WIDTH + tileSize - 1) / tileSize  + 1, (IMG_HEIGHT + tileSize - 1) / tileSize + 1);
	dim3 block_size(dimSize, dimSize);
	/*GrayPicMultipleBlocks << <grid_size, dim3(dimSize, dimSize) >> > ();*/

	
	hipDeviceSynchronize();

	AnyFilter<<<grid_size,block_size>>>();


	hipDeviceSynchronize();

	hipMemcpyFromSymbol(host_image, HIP_SYMBOL(dev_img_result), IMG_HEADER + IMG_WIDTH * IMG_HEIGHT * Channel * sizeof(unsigned char));
	hipDeviceSynchronize();



	fopen_s(&f_output_img, IMG_OUTPUT, "wb");
	fwrite(host_image, 1, IMG_HEADER + IMG_WIDTH * IMG_HEIGHT * Channel, f_output_img);
	fclose(f_output_img);
	free(img);
	free(host_image);

}
